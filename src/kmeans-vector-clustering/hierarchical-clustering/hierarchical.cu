#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include "kmeans.h"
#include "lsh.h"

__device__ float get_distance(const float* point, const float* centroid, int DIM)
{
    float squaredDistance = 0.0f;
    for (int d = 0; d < DIM; ++d)
    {
        float vecDiff = point[d] - centroid[d];
        squaredDistance += vecDiff * vecDiff;
    }
    return sqrt(squaredDistance);
}

__global__ void kmeans_labeling_kernel(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int N, int K, int DIM)
{
    const int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index of each thread, representing the position of the data point in the array
    if (globalThreadIndex >= N) return;

    float minDistance = INFINITY;
    int closestCenterIndex = -1;

    const float* curPoint = &d_samples[globalThreadIndex * DIM];

    // Iterate through cluster centroids to calculate distance
    for (int k = 0; k < K; ++k)
    {
        const float* curCentroid = &d_clusterCenters[k * DIM];
        float distance = get_distance(curPoint, curCentroid, DIM);
        if (distance < minDistance)
        {
            minDistance = distance;
            closestCenterIndex = k;
        }
    }

    d_clusterIndices[globalThreadIndex] = closestCenterIndex;
}

__global__ void kmeans_update_centers_kernel(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int *d_clusterSizes, int N, int K, int DIM)
{
    const int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIndex >= N) return;

    int cluster_id = d_clusterIndices[globalThreadIndex];

    // Accumulate the coordinates of each data point to the cluster centroid it belongs to
    for (int d = 0; d < DIM; ++d)
    {
        atomicAdd(&d_clusterCenters[cluster_id * DIM + d], d_samples[globalThreadIndex * DIM + d]);
    }

    // Increase the count of data points included in the cluster the current data point belongs to by 1
    atomicAdd(&d_clusterSizes[cluster_id], 1);
}

__global__ void kmeans_average_centers_kernel(float *d_clusterCenters, int *d_clusterSizes, int K, int DIM)
{
    const int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIndex >= K) return;

    // Calculate the new centroid of each cluster by computing the average coordinates 
    for (int d = 0; d < DIM; ++d)
    {
        if (d_clusterSizes[globalThreadIndex] > 0) {
            d_clusterCenters[globalThreadIndex * DIM + d] /= d_clusterSizes[globalThreadIndex];
        }
    }
}

void launch_kmeans_labeling(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int N, int TPB, int K, int DIM)
{
    kmeans_labeling_kernel<<<(N + TPB - 1) / TPB, TPB>>>(d_samples, d_clusterIndices, d_clusterCenters, N, K, DIM);
}

void launch_kmeans_update_center(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int *d_clusterSizes, int N, int TPB, int K, int DIM)
{
    hipMemset(d_clusterCenters, 0, K * DIM * sizeof(float));
    hipMemset(d_clusterSizes, 0, K * sizeof(int));

    kmeans_update_centers_kernel<<<(N + TPB - 1) / TPB, TPB>>>(d_samples, d_clusterIndices, d_clusterCenters, d_clusterSizes, N, K, DIM);
    kmeans_average_centers_kernel<<<(K + TPB - 1) / TPB, TPB>>>(d_clusterCenters, d_clusterSizes, K, DIM);
}

__global__ void lsh_coarse_cluster_kernel(float* d_data, int* d_coarse_labels, const float* d_random_proj, int N, int DIM, int num_buckets) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N) return;
    float dot = 0.0f;
    for (int d = 0; d < DIM; ++d) dot += d_data[idx * DIM + d] * d_random_proj[d];
    float normalized = fminf(fmaxf((dot + 10.0f) / 20.0f, 0.0f), 1.0f);
    int bucket = (int)(normalized * num_buckets) % num_buckets;
    d_coarse_labels[idx] = bucket;
}

void launch_lsh(float* d_data, int* d_coarse_labels, const float* d_random_proj, int N, int DIM, int num_buckets) {
    dim3 block(256), grid((N + block.x - 1) / block.x);
    lsh_coarse_cluster_kernel<<<grid, block>>>(d_data, d_coarse_labels, d_random_proj, N, DIM, num_buckets);
    hipDeviceSynchronize();
}