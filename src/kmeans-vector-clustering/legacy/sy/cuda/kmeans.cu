#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include "include/kmeans.h"

__device__ float get_distance(const float* point, const float* centroid, int DIM)
{
    float squaredDistance = 0.0f;
    for (int d = 0; d < DIM; ++d)
    {
        float vecDiff = point[d] - centroid[d];
        squaredDistance += vecDiff * vecDiff;
    }
    return sqrt(squaredDistance);
}

__global__ void kmeans_labeling_kernel(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int N, int K, int DIM)
{
    const int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index of each thread, representing the position of the data point in the array
    if (globalThreadIndex >= N) return;

    float minDistance = INFINITY;
    int closestCenterIndex = -1;

    const float* curPoint = &d_samples[globalThreadIndex * DIM];

    // Iterate through cluster centroids to calculate distance
    for (int k = 0; k < K; ++k)
    {
        const float* curCentroid = &d_clusterCenters[k * DIM];
        float distance = get_distance(curPoint, curCentroid, DIM);
        if (distance < minDistance)
        {
            minDistance = distance;
            closestCenterIndex = k;
        }
    }

    d_clusterIndices[globalThreadIndex] = closestCenterIndex;
}

__global__ void kmeans_update_centers_kernel(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int *d_clusterSizes, int N, int K, int DIM)
{
    const int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIndex >= N) return;

    int cluster_id = d_clusterIndices[globalThreadIndex];

    // Accumulate the coordinates of each data point to the cluster centroid it belongs to
    for (int d = 0; d < DIM; ++d)
    {
        atomicAdd(&d_clusterCenters[cluster_id * DIM + d], d_samples[globalThreadIndex * DIM + d]);
    }

    // Increase the count of data points included in the cluster the current data point belongs to by 1
    atomicAdd(&d_clusterSizes[cluster_id], 1);
}

__global__ void kmeans_average_centers_kernel(float *d_clusterCenters, int *d_clusterSizes, int K, int DIM)
{
    const int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIndex >= K) return;

    // Calculate the new centroid of each cluster by computing the average coordinates 
    for (int d = 0; d < DIM; ++d)
    {
        if (d_clusterSizes[globalThreadIndex] > 0) {
            d_clusterCenters[globalThreadIndex * DIM + d] /= d_clusterSizes[globalThreadIndex];
        }
    }
}

void launch_kmeans_labeling(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int N, int TPB, int K, int DIM)
{
    kmeans_labeling_kernel<<<(N + TPB - 1) / TPB, TPB>>>(d_samples, d_clusterIndices, d_clusterCenters, N, K, DIM);
}

void launch_kmeans_update_center(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int *d_clusterSizes, int N, int TPB, int K, int DIM)
{
    hipMemset(d_clusterCenters, 0, K * DIM * sizeof(float));
    hipMemset(d_clusterSizes, 0, K * sizeof(int));

    kmeans_update_centers_kernel<<<(N + TPB - 1) / TPB, TPB>>>(d_samples, d_clusterIndices, d_clusterCenters, d_clusterSizes, N, K, DIM);
    kmeans_average_centers_kernel<<<(K + TPB - 1) / TPB, TPB>>>(d_clusterCenters, d_clusterSizes, K, DIM);
}
