#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <device_launch_parameter.h>
#include <math.h>
#include "../include/kmeans_bank.h"
#define WARPSIZE 32

__device__ int d_chunkSize = 256;

__global__ void kmeans_labeling_kernel(
    float* d_samples,
    int* d_clusterIndices,
    float* d_clusterCenters,
    int *d_clusterSizes,
    int sharedMemSize,
    int N, int K, int DIM
) {

    extern __shared__ float sharedMemory[];
    float *squaredDistanceMem = &sharedMemory[0];
    int partialDim = DIM / WARPSIZE;
    int distanceSize = (partialDim + 1) * WARPSIZE;
    float *sharedCentroids = &sharedMemory[distanceSize];

    const int tid = threadIdx.x;
    squaredDistanceMem[DIM] = INFINITY;
    int *cluster_id = (int *)&squaredDistanceMem[DIM + 1];

    int maxCentroidSapce = sharedMemSize - (distanceSize * sizeof(float));
    int maxk = maxCentroidSapce / DIM / sizeof(float);
    int k = maxk;
    for(int o=0; o<K; o+=k)
    {
        k = min(maxk, K-o);

        for(int j=tid; j<k*DIM; j+=blockDim.x)
        {
            sharedCentroids[j] = d_clusterCenters[o * DIM + j];
        }
        __syncthreads();

        for(int j=0; j<k; j++)
        {
            for(int i=tid; i<DIM; i+=blockDim.x)
            {
                float dataPoint = d_samples[blockIdx.x * DIM + i];
                float kPoint = sharedCentroids[j * DIM + i];
                float diff = dataPoint - kPoint;
                squaredDistanceMem[i] = diff * diff;
            }

            for( int stride = distanceSize/2; stride>0; stride/=2)
            {
                for(int i=tid; i<stride; i+=blockDim.x)
                {
                    if (i+stride<DIM)
                    {
                        squaredDistanceMem[i] += squaredDistanceMem[i + stride];
                    }
                }
                __syncthreads();
            }

            if(tid==0)
            {
                if (squaredDistanceMem[DIM] > squaredDistanceMem[tid])
                {
                    squaredDistanceMem[DIM] = squaredDistanceMem[tid];
                    *cluster_id = o + j;
                }
            }
        }
    }

    if(tid==0)
    {
        atomicAdd(&d_clusterSizes[*cluster_id], 1);
        d_clusterIndices[blockIdx.x] = *cluster_id;
    }
}


__global__ void kmeans_update_centers_kernel(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int *d_clusterSizes, int N, int K, int DIM)
{
    // extern __shared__ float sharedMemory[];
    // float *pointSum = &sharedMemory[0];
    // int pointNum = blockIdx.x;
    // int cluster_id = d_clusterIndices[pointNum];
    // const int tid = threadIdx.x;

    // // Accumulate the coordinates of each data point to the cluster centroid it belongs to
    // // if (tid == 0)
    // // {
    // //     atomicAdd(&d_clusterSizes[cluster_id], 1);
    // // }

    // for (int i = tid; i < DIM; i += blockDim.x)
    // {
    //     pointSum[i] = d_samples[pointNum * DIM + i]; 
    // }
    // __syncthreads();

    // for (int i = tid; i < DIM; i += blockDim.x)
    // {
    //     atomicAdd(&d_clusterCenters[cluster_id * DIM + i], (pointSum[i] / d_clusterSizes[cluster_id]));
    // }

    // __syncthreads();

    // === fix ===
    int pointNum = blockIdx.x;
    int tid      = threadIdx.x;
    if (pointNum >= N) return;

    int cid = d_clusterIndices[pointNum];
    int csz = d_clusterSizes[cid];
    if (csz <= 0) return;

    const float *x = d_samples + (size_t)pointNum * DIM;
    float inv = 1.0f / (float)csz;

    for (int i = tid; i < DIM; i += blockDim.x) 
    {
        atomicAdd(&d_clusterCenters[(size_t)cid * DIM + i], x[i] * inv);
    }
}


__global__ void kmeans_average_centers_kernel(float *d_clusterCenters, int *d_clusterSizes, int K, int DIM)
{
    const int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIndex >= K) return;

    // Calculate the new centroid of each cluster by computing the average coordinates 
    for (int d = 0; d < DIM; ++d)
    {
        if (d_clusterSizes[globalThreadIndex] > 0) {
            d_clusterCenters[globalThreadIndex * DIM + d] /= d_clusterSizes[globalThreadIndex];
        }
    }
}

void launch_kmeans_labeling(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int *d_clusterSizes, int N, int TPB, int K, int DIM)
{
    int shredMemSize = 8 * sizeof(float) * DIM;
    hipMemset(d_clusterSizes, 0, K * sizeof(int));
    kmeans_labeling_kernel<<<N, TPB, shredMemSize>>>(d_samples, d_clusterIndices, d_clusterCenters, d_clusterSizes, shredMemSize,  N, K, DIM);
}

void launch_kmeans_update_center(float *d_samples, int *d_clusterIndices, float *d_clusterCenters, int *d_clusterSizes, int N, int TPB, int K, int DIM)
{
    hipMemset(d_clusterCenters, 0, K * DIM * sizeof(float));
    int sharedMemSize = sizeof(float) * DIM;

    kmeans_update_centers_kernel<<<N, TPB, sharedMemSize>>>(d_samples, d_clusterIndices, d_clusterCenters, d_clusterSizes, N, K, DIM);
}

