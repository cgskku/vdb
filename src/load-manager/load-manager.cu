#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include "load-manager.h"
#include <stdio.h>

__device__ float get_distance_l1(const float* point, const float* centroid, int DIM)
{
    float dist = 0.0f;
    for (int d = 0; d < DIM; ++d)
        dist += fabsf(point[d] - centroid[d]);
    return dist;
}

__device__ float get_distance_l2(const float* point, const float* centroid, int DIM)
{
    float squaredDistance = 0.0f;
    for (int d = 0; d < DIM; ++d) {
        float vecDiff = point[d] - centroid[d];
        squaredDistance += vecDiff * vecDiff;
    }
    return sqrtf(squaredDistance);
}

__device__ float get_distance_cosine(const float* point, const float* centroid, int DIM)
{
    float dot = 0.0f, norm_point = 0.0f, norm_centroid = 0.0f;
    for (int d = 0; d < DIM; ++d) {
        dot += point[d] * centroid[d];
        norm_point += point[d] * point[d];
        norm_centroid += centroid[d] * centroid[d];
    }
    float denom = sqrtf(norm_point) * sqrtf(norm_centroid) + 0.00000001f;
    float cosine_similarity = dot / denom;

    return 1.0f - cosine_similarity;
}

__global__ void distance_l1_kernel(const float* db_vectors, const float* query, float* dists, int N, int DIM) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N) return;
    dists[idx] = get_distance_l1(db_vectors + idx * DIM, query, DIM);
}

__global__ void distance_l2_kernel(const float* db_vectors, const float* query, float* dists, int N, int DIM) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N) return;
    dists[idx] = get_distance_l2(db_vectors + idx * DIM, query, DIM);
}

__global__ void distance_cosine_kernel(const float* db_vectors, const float* query, float* dists, int N, int DIM) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N) return;
    dists[idx] = get_distance_cosine(db_vectors + idx * DIM, query, DIM);
}

void launch_distance_kernel(const float *d_db_vectors, const float *d_query, float *d_dists,
                           int N, int DIM, NormType normType, int TPB)
{
    int numBlocks = (N + TPB - 1) / TPB;
    if (normType == L1_NORM)
        distance_l1_kernel<<<numBlocks, TPB>>>(d_db_vectors, d_query, d_dists, N, DIM);
    else if (normType == L2_NORM)
        distance_l2_kernel<<<numBlocks, TPB>>>(d_db_vectors, d_query, d_dists, N, DIM);
    else
        distance_cosine_kernel<<<numBlocks, TPB>>>(d_db_vectors, d_query, d_dists, N, DIM);
}

__device__ float get_distance_l2_pair(const float* a, const float* b, int DIM) {
    float dist = 0.0f;
    for (int d = 0; d < DIM; ++d) {
        float diff = a[d] - b[d];
        dist += diff * diff;
    }
    return sqrtf(dist);
}

__global__ void pairwise_distance_kernel(const float* db_vectors, float* dists, int N, int DIM) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < N) {
        size_t idx = static_cast<size_t>(i) * N + j;
        if (idx >= static_cast<size_t>(N) * static_cast<size_t>(N)) {
            printf("Index out of bound: i=%d j=%d idx=%zu\n", i, j, idx);
        }
        dists[idx] = get_distance_l2_pair(db_vectors + i * DIM, db_vectors + j * DIM, DIM);
    }
}

void launch_pairwise_distance_kernel(const float *d_db_vectors, float *d_pairwise, int N, int DIM, NormType normType, int blockX, int blockY) {
    dim3 block(blockX, blockY);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    pairwise_distance_kernel<<<grid, block>>>(d_db_vectors, d_pairwise, N, DIM);
}

__global__ void pairwise_distance_tile_kernel(const float* db_vectors, const float* db_vectors2, float* tile_out, int N, int DIM, int row_offset, int col_offset, int tile_rows, int tile_cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < tile_rows && col < tile_cols) {
        int idx_i = row_offset + row;
        int idx_j = col_offset + col;
        tile_out[row * tile_cols + col] =
            get_distance_l2(db_vectors + idx_i * DIM, db_vectors2 + idx_j * DIM, DIM);
    }
}

__global__ void pairwise_distance_tile_kernel_transpose(
    const float* db_vectors,      // [N x DIM], row-major
    const float* db_vectors2_T,   // [DIM x tile_cols], col-major
    float* tile_out,
    int N, int DIM,
    int row_offset, int col_offset, int tile_rows, int tile_cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < tile_rows && col < tile_cols) {
        int idx_i = row_offset + row;
        int idx_j = col;
        float dist = 0.0f;
        for (int d = 0; d < DIM; ++d) {
            float a = db_vectors[idx_i * DIM + d];          // row vector
            float b = db_vectors2_T[d * tile_cols + idx_j]; // col vector
            float diff = a - b;
            dist += diff * diff;
        }
        tile_out[row * tile_cols + col] = sqrtf(dist);
    }
}

void launch_pairwise_distance_tile_kernel(
    const float* d_db_vectors, const float* d_db_vectors2,
    float* d_tile,
    int N, int DIM,
    int row_offset, int col_offset, int tile_rows, int tile_cols,
    dim3 block, dim3 grid)
{
    pairwise_distance_tile_kernel<<<grid, block>>>(
        d_db_vectors, d_db_vectors2, d_tile, N, DIM,
        row_offset, col_offset, tile_rows, tile_cols);
}

void launch_pairwise_distance_tile_kernel_transpose(
    const float* d_db_vectors, const float* d_db_vectors2_T,
    float* d_tile,
    int N, int DIM,
    int row_offset, int tile_rows, int tile_cols,
    dim3 block, dim3 grid)
{
    pairwise_distance_tile_kernel_transpose<<<grid, block>>>(
        d_db_vectors, d_db_vectors2_T, d_tile, N, DIM,
        row_offset, 0, tile_rows, tile_cols);
}